#include "hip/hip_runtime.h"
#include<iostream>
#include "mult.cuh"
#define BLOCK_SIZE 2

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void custom_cudaMalloc(void** devPtr, size_t size)
{
    hipMalloc(devPtr, size);
}

void custom_cudaFree ( void* devPtr )
{
    hipFree(devPtr);
}

void custom_cudaMemcpy_d2h ( void* dst, const void* src, size_t count)
{
    hipMemcpy(dst, src, count, hipMemcpyDeviceToHost);
}

void custom_cudaMemcpy_h2d ( void* dst, const void* src, size_t count)
{
    hipMemcpy(dst, src, count, hipMemcpyHostToDevice);
}

__global__ void hello()
{
    printf("Hi from GPU %d %d\n", threadIdx.x, blockIdx.x);
}

void call_cuda()
{
	hello<<<2,32>>>();
	hipDeviceSynchronize();
}


/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */
__global__ void MatrixMulCUDA(float *C, const float *A,
    const float *B, int wA,
    int wB) {

  // Block index
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Index of the first sub-matrix of A processed by the block
  int aBegin = wA * BLOCK_SIZE * by;

  // Index of the last sub-matrix of A processed by the block
  int aEnd   = aBegin + wA - 1;

  // Step size used to iterate through the sub-matrices of A
  int aStep  = BLOCK_SIZE;

  // Index of the first sub-matrix of B processed by the block
  int bBegin = BLOCK_SIZE * bx;

  // Step size used to iterate through the sub-matrices of B
  int bStep  = BLOCK_SIZE * wB;

  // Csub is used to store the element of the block sub-matrix
  // that is computed by the thread
  float Csub = 0;

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (int a = aBegin, b = bBegin;
       a <= aEnd;
       a += aStep, b += bStep) {
    // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Load the matrices from device memory
    // to shared memory; each thread loads
    // one element of each matrix
    As[ty][tx] = A[a + wA * ty + tx];
    Bs[ty][tx] = B[b + wB * ty + tx];

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Multiply the two matrices together;
    // each thread computes one element
    // of the block sub-matrix
#pragma unroll

    for (int k = 0; k < BLOCK_SIZE; ++k) {
      Csub += As[ty][k] * Bs[k][tx];
    }

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write the block sub-matrix to device memory;
  // each thread writes one element
  int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
  C[c + wB * ty + tx] = Csub;
}

void computeMM(const float *A, const float *B, float *C, int m, int k, int n)
{
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((m+threads.x-1)/threads.x, (n+threads.y-1)/threads.y); 
    //compute_tf32gemm_async_copy(A, B, C);
    MatrixMulCUDA<<<grid, threads>>>(C, A, B, k, n);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    return;
}


// Performs an MxNxK tf32 GEMM (C=alpha*A*B + beta*C) assuming:
//  1) Matrices are packed in memory.
//  2) M, N and K are multiples of 16, 16 and 8 respectively. 
//  3) A is row major, B is column major matrix.
// Note: This is a less performant version of the compute_tf32gemm kernel. It is designed for
//       demonstration purposes only to show the CUDA WMMA API use without relying on
//       availability of the shared memory.
__global__ void simple_wmma_tf32gemm(float *a, float *b,  float *d, int m_ld, int n_ld, int k_ld)
{
#if __CUDA_ARCH__ >= 800
   // Leading dimensions. Packed with no transpositions.
    int lda = k_ld;
    int ldb = k_ld;
    int ldc = n_ld;

   // Tile using a 2D grid
   int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
   int warpN = (blockIdx.y * blockDim.y + threadIdx.y);
 
   // Declare the fragments
   wmma::fragment<wmma::matrix_a, M, N, K, wmma::precision::tf32, wmma::row_major> a_frag;
   wmma::fragment<wmma::matrix_b, M, N, K, wmma::precision::tf32, wmma::col_major> b_frag;
   wmma::fragment<wmma::accumulator, M, N, K, float> acc_frag;
   wmma::fragment<wmma::accumulator, M, N, K, float> c_frag;

   wmma::fill_fragment(acc_frag, 0.0f);

   // Loop over k
   for (int i = 0; i < k_ld; i += K) {
      int aCol = i; 
      int aRow = warpM * M;

      //int bCol = i;
      //int bRow = warpN * N;
      int bCol = warpN * N;
      int bRow = i;

      // Bounds checking
      if (aRow < m_ld && aCol < k_ld && bRow < k_ld && bCol < n_ld) {
         // Load the inputs
         wmma::load_matrix_sync(a_frag, a + aCol + aRow * lda, lda);
         wmma::load_matrix_sync(b_frag, b + bRow + bCol * ldb, ldb);
 
 #pragma unroll
        for (int t = 0; t < a_frag.num_elements; t++) {
                a_frag.x[t] =  wmma::__float_to_tf32(a_frag.x[t]);
        }

 #pragma unroll
        for (int t = 0; t < b_frag.num_elements; t++) {
                b_frag.x[t] =  wmma::__float_to_tf32(b_frag.x[t]);
        }
         // Perform the matrix multiplication
         wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);

      }
   }

   // Load in the current value of c, scale it by beta, and add this our result scaled by alpha
   int cCol = warpN * N;
   int cRow = warpM * M;

   if (cRow < m_ld && cCol < n_ld) {
      //wmma::load_matrix_sync(c_frag, c + cCol + cRow * ldc, ldc, wmma::mem_row_major);

      for(int i=0; i < c_frag.num_elements; i++) {
         c_frag.x[i] = acc_frag.x[i] + c_frag.x[i];
      }

      // Store the output
      wmma::store_matrix_sync(d + cCol + cRow * ldc, c_frag, ldc, wmma::mem_row_major);
   }
#endif
}

__global__ void compute_tf32gemm_async_copy(const float *A, const float *B, const float *C, float *D, const float alpha, float beta)
{
#if __CUDA_ARCH__ >= 800
    extern __shared__ float shmem[][CHUNK_K * K + SKEW_FLOAT];

    // Warp and lane identification.
    const unsigned int warpId = threadIdx.x / WARP_SIZE;
    const unsigned int laneId = threadIdx.x % WARP_SIZE;

    // This pointer is used to access the C and D matrix tiles this warp computes.
    float *shmem_warp_tile_ptr = (float*)&shmem[0][0] + (warpId / BLOCK_ROW_WARPS) * SHMEM_STRIDE * N * BLOCK_ROW_WARPS + (warpId % BLOCK_ROW_WARPS) * SHMEM_OFFSET;

    // This pointer is used to stream the C and D matrices block-wide tile to and from shared memory.
    float *shmem_warp_stream_ptr = (float*)&shmem[0][0] + warpId * SHMEM_STRIDE * N;

    // Offset in shared memory from which the B matrix is stored.
    constexpr size_t shmem_idx_b_off = BLOCK_COL_TILES * M;

    // Adjust the beta scaler, as it'll be multiplied by alpha at the end of
    // each tile computation. Technically this is not generally correct (may result
    // in a loss of precision). Zero still needs to be specially handled though.
    beta /= alpha;

    cuda::pipeline<cuda::thread_scope_thread> pipe = cuda::make_pipeline();
    const auto shape4 = cuda::aligned_size_t<alignof(float4)>(sizeof(float4));
    constexpr int loadStride = 2; // load 4 floats, so left-shift by 2.

    // Each CTA slides along the 128 x 128 tiles from the top left corner of the matrix to the
    // right and down, and selects the next tile to compute. Once there's no such tile,
    // all warps in this CTA exit.
    for(unsigned int block_pos = blockIdx.x;; block_pos += gridDim.x) {
        const unsigned int block_tile_i = ((block_pos * BLOCK_ROW_TILES) / N_TILES) * (BLOCK_COL_TILES);
        const unsigned int block_tile_j = (block_pos * BLOCK_COL_TILES) % N_TILES;

        // Stop when there are no more D matrix tiles to compute in this CTA.
        if (block_tile_i >= M_TILES) {
            break;
        }

        // This warp's pointer to the C matrix data to copy memory from to shared memory.
        const size_t gmem_idx = (block_tile_i + warpId) * M * GLOBAL_MEM_STRIDE + block_tile_j * N;
        const float *src_gmem_warp_stream_ptr = &C[gmem_idx];

        // Stream multiple C tiles to shared memory.
#pragma unroll
        for (int i = 0; i < N; i++) {
            pipe.producer_acquire();
            cuda::memcpy_async(&shmem_warp_stream_ptr[(SHMEM_STRIDE * i) + (laneId << loadStride)],
                                &src_gmem_warp_stream_ptr[(GLOBAL_MEM_STRIDE * i) + (laneId << loadStride)],
                                shape4, pipe);
            pipe.producer_commit();
        }
        // Now wait for all the above issued 8 batches to complete.
        cuda::pipeline_consumer_wait_prior<0>(pipe);
        __syncthreads();

        // These fragments will accumulate the result of A and B matrix fragment multiplications
        // along the K_GLOBAL dimension.
        wmma::fragment<wmma::accumulator, M, N, K, float> c[WARP_COL_TILES][WARP_ROW_TILES];

        // Load the C matrix tiles into fragments from shared memory.
#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
            for (int j = 0; j < WARP_ROW_TILES; j++) {
                const float *tile_ptr = shmem_warp_tile_ptr + i * SHMEM_STRIDE * N + j * N;

                wmma::load_matrix_sync(c[i][j], tile_ptr, SHMEM_STRIDE, C_LAYOUT);
                // Scale the C matrix.
#pragma unroll
                for (int t = 0; t < c[i][j].num_elements; t++) {
                    c[i][j].x[t] *= beta;
                }
            }
        }
        pipe.consumer_release();

        // sync here so that shared memory can then be used for loading A & B matrices.
        __syncthreads();

        // Select what warp copies what matrix to shared memory.
        // Warps 0-3 copy the A matrix, warps 4-7 copy the B matrix.
        const float *warp_ptr = (warpId < (WARPS_PER_BLOCK/2)) ? (&A[block_tile_i * M * K_GLOBAL] + M * K_GLOBAL * (warpId % (WARPS_PER_BLOCK/2)) * 2) :
                                              (&B[block_tile_j * N * K_GLOBAL] + N * K_GLOBAL * (warpId % (WARPS_PER_BLOCK/2)) * 2);

        constexpr int chunksPerLane = ((WARP_SIZE/2) / CHUNK_COPY_LINES_PER_WARP) * 2;
        const int laneLoadElem = (laneId % CHUNK_COPY_LINE_LANES) << loadStride;
        const int stridePerLaneCopy = (laneId / CHUNK_COPY_LINE_LANES);
        // Go through the global K dimension by a fixed step at a time.
#pragma unroll
        for (int tile_k = 0; tile_k < K_TILES; tile_k += CHUNK_K) {
            // Copy slices of the A and B matrices to shared memory.
            // The first half of the warps in the CTA copy the A matrix, the rest copy the B matrix.
            // As for tf32 MMA  M == N we use M for warp 4-7 + shmem_idx_b_off.
            size_t shmem_idx =  (M * (warpId % (WARPS_PER_BLOCK/2)) * 2)  + ((warpId / (WARPS_PER_BLOCK/2)) * shmem_idx_b_off);
            // First half of the warp copies the first row / column of the matrix,
            // the second half of the warp copies the next.
            const float *lane_ptr = (warp_ptr + tile_k * K + stridePerLaneCopy * K_GLOBAL + laneLoadElem);

            // Shift the second half of the warp to the next row / column in the shared memory.
            shmem_idx += stridePerLaneCopy;

#pragma unroll
            for(int i = 0; i < chunksPerLane; i++) {
                // Copy 16 bytes at once in each lane.
                pipe.producer_acquire();
                cuda::memcpy_async(&shmem[shmem_idx][laneLoadElem], lane_ptr, shape4, pipe);
                pipe.producer_commit();

                // Advance the global memory pointer and the shared memory index.
                lane_ptr = lane_ptr + K_GLOBAL * CHUNK_COPY_LINES_PER_WARP;
                shmem_idx += CHUNK_COPY_LINES_PER_WARP;
            }

            cuda::pipeline_consumer_wait_prior<0>(pipe);
            __syncthreads();

            // Compute a grid of C matrix tiles in each warp.
#pragma unroll
            for (int k_step = 0; k_step < CHUNK_K; k_step++) {
                wmma::fragment<wmma::matrix_a, M, N, K, wmma::precision::tf32, wmma::row_major> a[WARP_COL_TILES];
                wmma::fragment<wmma::matrix_b, M, N, K, wmma::precision::tf32, wmma::col_major> b[WARP_ROW_TILES];

#pragma unroll
                for (int i = 0; i < WARP_COL_TILES; i++) {
                    size_t shmem_idx_a = (warpId / BLOCK_ROW_WARPS) * M * BLOCK_ROW_WARPS + (i * M);
                    const float *tile_ptr = &shmem[shmem_idx_a][k_step * K];

                    wmma::load_matrix_sync(a[i], tile_ptr, K * CHUNK_K + SKEW_FLOAT);

#pragma unroll
                    for (int t = 0; t < a[i].num_elements; t++) {
                        a[i].x[t] = wmma::__float_to_tf32(a[i].x[t]);
                    }
#pragma unroll
                    for (int j = 0; j < WARP_ROW_TILES; j++) {
                        if (i == 0) {
                            // Load the B matrix fragment once, because it is going to be reused
                            // against the other A matrix fragments.
                            size_t shmem_idx_b = shmem_idx_b_off + (WARP_ROW_TILES * N) * (warpId%2) + (j * N);
                            const float *tile_ptr = &shmem[shmem_idx_b][k_step * K];

                            wmma::load_matrix_sync(b[j], tile_ptr, K * CHUNK_K + SKEW_FLOAT);
#pragma unroll
                            for (int t = 0; t < b[j].num_elements; t++) {
                                b[j].x[t] =  wmma::__float_to_tf32(b[j].x[t]);
                            }
                        }

                        wmma::mma_sync(c[i][j], a[i], b[j], c[i][j]);
                    }
                }
            }
            pipe.consumer_release();
            __syncthreads();
        }

        // Store the D fragments to shared memory.
#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
            for (int j = 0; j < WARP_ROW_TILES; j++) {
#pragma unroll
                // Uniform, point-wise transformations of ALL fragment elements by ALL threads in the
                // warp are well-defined even though element indices within fragment storage are not defined.
                for (int t = 0; t < c[i][j].num_elements; t++)
                    c[i][j].x[t] *= alpha;

                float *tile_ptr = shmem_warp_tile_ptr + i * SHMEM_STRIDE * N + j * N;

                wmma::store_matrix_sync(tile_ptr, c[i][j], SHMEM_STRIDE, C_LAYOUT);
            }
        }

        __syncthreads();

        // Now that shared memory contains all the D tiles, stream them to global memory.
        float *dst_gmem_warp_stream_ptr = &D[gmem_idx];

#pragma unroll
        for (int i = 0; i < N; i++) {
            *((float4*)(dst_gmem_warp_stream_ptr + GLOBAL_MEM_STRIDE * i) + laneId) =
                *((float4*)(shmem_warp_stream_ptr + SHMEM_STRIDE * i) + laneId);
        }

        __syncthreads();
    }
#endif
}
