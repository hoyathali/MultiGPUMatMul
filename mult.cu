#include "hip/hip_runtime.h"
#include<iostream>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <cuda/pipeline>

#include "mult.cuh"

#define BLOCK_SIZE 2

#ifndef MAX
#define MAX(a, b) (a > b ? a : b)
#endif

#ifndef SHARED_MEMORY_LIMIT_64K
// Set this to 0 to use more than 64 Kb of shared memory to cache data, to
// improve the performance of the computations on GPU.
// Note that you need a GPU that can have more than 64 Kb of shared memory
// per multiprocessor.
#define SHARED_MEMORY_LIMIT_64K 0
#endif

#define WARP_SIZE 32

// M, N and K represent the wmma tile size
// These values should not be modified
#define M 16
#define N 16
#define K 8

#define M_TILES (BAND_SIZE/M)
#define N_TILES (BAND_SIZE/N)
#define K_TILES (K_GLOBAL/K)

#define C_LAYOUT wmma::mem_row_major

#define WARPS_PER_BLOCK 8
#define THREADS_PER_BLOCK (WARP_SIZE * WARPS_PER_BLOCK)

#if SHARED_MEMORY_LIMIT_64K
// With only 64 Kb shared memory available, we can fit two 8-tile chunks of
// the A and B matrix data, that is (M = 16) * (K = 8) * 8 * (CHUNK_K = 8)
// * sizeof(float) = 32 Kb each.
// (i.e. two 8x8 arrays of tiles of 16x8 float-typed elements per CTA).
// But we cannot account the 8 Kb total skew overhead, without which the performance
// would be severely impacted. So we choose to reduce the chunk size in half,
// i.e. the amount of A and B matrix data we cache in shared memory.
// Accordingly, this doubles the number of outer iterations across the global K
// dimension, which only slightly impacts the performance.
#define CHUNK_K 4
#else
#define CHUNK_K 8
#endif

#define CHUNK_LINE_BYTES (CHUNK_K * K * sizeof(float))
#define WARP_COPY_BYTES (WARP_SIZE * sizeof(int4))
#define CHUNK_COPY_LINES_PER_WARP (WARP_COPY_BYTES / CHUNK_LINE_BYTES)
#define CHUNK_COPY_LINE_LANES (WARP_SIZE / CHUNK_COPY_LINES_PER_WARP)

#define BLOCK_ROW_WARPS 2
#define BLOCK_COL_WARPS 4

#define WARP_ROW_TILES 4
#define WARP_COL_TILES 2

#define BLOCK_ROW_TILES (WARP_ROW_TILES * BLOCK_ROW_WARPS)
#define BLOCK_COL_TILES (WARP_COL_TILES * BLOCK_COL_WARPS)

#define GLOBAL_MEM_STRIDE (N * N_TILES)

#define SHMEM_STRIDE (N * BLOCK_ROW_TILES)
#define SHMEM_OFFSET (N * WARP_ROW_TILES)

// The macro below is used to shift rows of the A matrix and columns of the B matrix
// in shared memory to minimize possible bank conflicts.
// Before performing the nvcuda::wmma::mma_sync operation, the warp must load the matrix
// data using the nvcuda::wmma::load_matrix_sync operation. Although the memory access pattern
// is not specified for that function, each lane in the warp can read one or multiple matrix
// elements from different matrix rows or columns.
// For shared memory, such access can result in bank conflicts if different rows / columns
// of the matrix map to the same bank. By shifting each row and column by a few bytes, we
// make sure that they map to different banks, thus reducing the number of possible bank
// conflicts.
// The number of 8 four-byte "float" elements is chosen as the minimum possible shift because
// we must keep each row and column 256-bit aligned, as required by nvcuda::wmma::load_matrix_sync.
#define SKEW_FLOAT 8

using namespace nvcuda;

hipDeviceProp_t deviceProp;

void custom_cudaMalloc(void** devPtr, size_t size)
{
    hipMallocManaged(devPtr, size);
}

void custom_cudaFree ( void* devPtr )
{
    hipFree(devPtr);
}

void custom_cudaMemcpy_d2h ( void* dst, const void* src, size_t count)
{
    hipMemcpy(dst, src, count, hipMemcpyDeviceToHost);
}

void custom_cudaMemcpy_h2d ( void* dst, const void* src, size_t count)
{
    hipMemcpy(dst, src, count, hipMemcpyHostToDevice);
}

__global__ void hello()
{
    printf("Hi from GPU %d %d\n", threadIdx.x, blockIdx.x);
}

void call_cuda()
{
	hello<<<2,32>>>();
	hipDeviceSynchronize();
}


/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */
__global__ void MatrixMulCUDA(float *C, const float *A,
    const float *B, int wA,
    int wB) {

  // Block index
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Index of the first sub-matrix of A processed by the block
  int aBegin = wA * BLOCK_SIZE * by;

  // Index of the last sub-matrix of A processed by the block
  int aEnd   = aBegin + wA - 1;

  // Step size used to iterate through the sub-matrices of A
  int aStep  = BLOCK_SIZE;

  // Index of the first sub-matrix of B processed by the block
  int bBegin = BLOCK_SIZE * bx;

  // Step size used to iterate through the sub-matrices of B
  int bStep  = BLOCK_SIZE * wB;

  // Csub is used to store the element of the block sub-matrix
  // that is computed by the thread
  float Csub = 0;

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (int a = aBegin, b = bBegin;
       a <= aEnd;
       a += aStep, b += bStep) {
    // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Load the matrices from device memory
    // to shared memory; each thread loads
    // one element of each matrix
    As[ty][tx] = A[a + wA * ty + tx];
    Bs[ty][tx] = B[b + wB * ty + tx];

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Multiply the two matrices together;
    // each thread computes one element
    // of the block sub-matrix
#pragma unroll

    for (int k = 0; k < BLOCK_SIZE; ++k) {
      Csub += As[ty][k] * Bs[k][tx];
    }

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write the block sub-matrix to device memory;
  // each thread writes one element
  int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
  C[c + wB * ty + tx] = Csub;
}


// Performs an MxNxK tf32 GEMM (C=alpha*A*B + beta*C) assuming:
//  1) Matrices are packed in memory.
//  2) M, N and K are multiples of 16, 16 and 8 respectively. 
//  3) A is row major, B is column major matrix.
// Note: This is a less performant version of the compute_tf32gemm kernel. It is designed for
//       demonstration purposes only to show the CUDA WMMA API use without relying on
//       availability of the shared memory.
__global__ void simple_wmma_tf32gemm(float *a, float *b,  float *d, int m_ld, int n_ld, int k_ld)
{
#if __CUDA_ARCH__ >= 800
   // Leading dimensions. Packed with no transpositions.
    int lda = k_ld;
    int ldb = k_ld;
    int ldc = n_ld;

   // Tile using a 2D grid
   int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
   int warpN = (blockIdx.y * blockDim.y + threadIdx.y);
 
   // Declare the fragments
   wmma::fragment<wmma::matrix_a, M, N, K, wmma::precision::tf32, wmma::row_major> a_frag;
   wmma::fragment<wmma::matrix_b, M, N, K, wmma::precision::tf32, wmma::col_major> b_frag;
   wmma::fragment<wmma::accumulator, M, N, K, float> acc_frag;
   wmma::fragment<wmma::accumulator, M, N, K, float> c_frag;

   wmma::fill_fragment(acc_frag, 0.0f);

   // Loop over k
   for (int i = 0; i < k_ld; i += K) {
      int aCol = i; 
      int aRow = warpM * M;

      //int bCol = i;
      //int bRow = warpN * N;
      int bCol = warpN * N;
      int bRow = i;

      // Bounds checking
      if (aRow < m_ld && aCol < k_ld && bRow < k_ld && bCol < n_ld) {
         // Load the inputs
         wmma::load_matrix_sync(a_frag, a + aCol + aRow * lda, lda);
         wmma::load_matrix_sync(b_frag, b + bRow + bCol * ldb, ldb);
 
 #pragma unroll
        for (int t = 0; t < a_frag.num_elements; t++) {
                a_frag.x[t] =  wmma::__float_to_tf32(a_frag.x[t]);
        }

 #pragma unroll
        for (int t = 0; t < b_frag.num_elements; t++) {
                b_frag.x[t] =  wmma::__float_to_tf32(b_frag.x[t]);
        }
         // Perform the matrix multiplication
         wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);

      }
   }

   // Load in the current value of c, scale it by beta, and add this our result scaled by alpha
   int cCol = warpN * N;
   int cRow = warpM * M;

   if (cRow < m_ld && cCol < n_ld) {
      //wmma::load_matrix_sync(c_frag, c + cCol + cRow * ldc, ldc, wmma::mem_row_major);

      for(int i=0; i < c_frag.num_elements; i++) {
         c_frag.x[i] = acc_frag.x[i] + c_frag.x[i];
      }

      // Store the output
      wmma::store_matrix_sync(d + cCol + cRow * ldc, c_frag, ldc, wmma::mem_row_major);
   }
#endif
}

__global__ void compute_tf32gemm_async_copy(const float *A, const float *B, float *D)
{
#if __CUDA_ARCH__ >= 800
    extern __shared__ float shmem[][CHUNK_K * K + SKEW_FLOAT];

    // Warp and lane identification.
    const unsigned int warpId = threadIdx.x / WARP_SIZE;
    const unsigned int laneId = threadIdx.x % WARP_SIZE;

    // This pointer is used to access the D matrix tiles this warp computes.
    float *shmem_warp_tile_ptr = (float*)&shmem[0][0] + (warpId / BLOCK_ROW_WARPS) * SHMEM_STRIDE * N * BLOCK_ROW_WARPS + (warpId % BLOCK_ROW_WARPS) * SHMEM_OFFSET;

    // This pointer is used to stream the D matrix block-wide tile to and from shared memory.
    float *shmem_warp_stream_ptr = (float*)&shmem[0][0] + warpId * SHMEM_STRIDE * N;

    // Offset in shared memory from which the B matrix is stored.
    constexpr size_t shmem_idx_b_off = BLOCK_COL_TILES * M;

    cuda::pipeline<cuda::thread_scope_thread> pipe = cuda::make_pipeline();
    const auto shape4 = cuda::aligned_size_t<alignof(float4)>(sizeof(float4));
    constexpr int loadStride = 2; // load 4 floats, so left-shift by 2.

    // Each CTA slides along the 128 x 128 tiles from the top left corner of the matrix to the
    // right and down, and selects the next tile to compute. Once there's no such tile,
    // all warps in this CTA exit.
    for(unsigned int block_pos = blockIdx.x;; block_pos += gridDim.x) {
        const unsigned int block_tile_i = ((block_pos * BLOCK_ROW_TILES) / N_TILES) * (BLOCK_COL_TILES);
        const unsigned int block_tile_j = (block_pos * BLOCK_COL_TILES) % N_TILES;

        // Stop when there are no more D matrix tiles to compute in this CTA.
        if (block_tile_i >= M_TILES) {
            break;
        }

        // This warp's pointer to the D matrix data to copy memory from to shared memory.
        const size_t gmem_idx = (block_tile_i + warpId) * M * GLOBAL_MEM_STRIDE + block_tile_j * N;

        // These fragments will accumulate the result of A and B matrix fragment multiplications
        // along the K_GLOBAL dimension.
        wmma::fragment<wmma::accumulator, M, N, K, float> c[WARP_COL_TILES][WARP_ROW_TILES];

        // Initialize the accumulator fragments to 0.
#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
            for (int j = 0; j < WARP_ROW_TILES; j++) {
		wmma::fill_fragment(c[i][j], 0.0f);
            }
        }

        // Select what warp copies what matrix to shared memory.
        // Warps 0-3 copy the A matrix, warps 4-7 copy the B matrix.
        const float *warp_ptr = (warpId < (WARPS_PER_BLOCK/2)) ? (&A[block_tile_i * M * K_GLOBAL] + M * K_GLOBAL * (warpId % (WARPS_PER_BLOCK/2)) * 2) :
                                              (&B[block_tile_j * N * K_GLOBAL] + N * K_GLOBAL * (warpId % (WARPS_PER_BLOCK/2)) * 2);

        constexpr int chunksPerLane = ((WARP_SIZE/2) / CHUNK_COPY_LINES_PER_WARP) * 2;
        const int laneLoadElem = (laneId % CHUNK_COPY_LINE_LANES) << loadStride;
        const int stridePerLaneCopy = (laneId / CHUNK_COPY_LINE_LANES);
        // Go through the global K dimension by a fixed step at a time.
#pragma unroll
        for (int tile_k = 0; tile_k < K_TILES; tile_k += CHUNK_K) {
            // Copy slices of the A and B matrices to shared memory.
            // The first half of the warps in the CTA copy the A matrix, the rest copy the B matrix.
            // As for tf32 MMA  M == N we use M for warp 4-7 + shmem_idx_b_off.
            size_t shmem_idx =  (M * (warpId % (WARPS_PER_BLOCK/2)) * 2)  + ((warpId / (WARPS_PER_BLOCK/2)) * shmem_idx_b_off);
            // First half of the warp copies the first row / column of the matrix,
            // the second half of the warp copies the next.
            const float *lane_ptr = (warp_ptr + tile_k * K + stridePerLaneCopy * K_GLOBAL + laneLoadElem);

            // Shift the second half of the warp to the next row / column in the shared memory.
            shmem_idx += stridePerLaneCopy;

#pragma unroll
            for(int i = 0; i < chunksPerLane; i++) {
                // Copy 16 bytes at once in each lane.
                pipe.producer_acquire();
                cuda::memcpy_async(&shmem[shmem_idx][laneLoadElem], lane_ptr, shape4, pipe);
                pipe.producer_commit();

                // Advance the global memory pointer and the shared memory index.
                lane_ptr = lane_ptr + K_GLOBAL * CHUNK_COPY_LINES_PER_WARP;
                shmem_idx += CHUNK_COPY_LINES_PER_WARP;
            }

            cuda::pipeline_consumer_wait_prior<0>(pipe);
            __syncthreads();

            // Compute a grid of D matrix tiles in each warp.
#pragma unroll
            for (int k_step = 0; k_step < CHUNK_K; k_step++) {
                wmma::fragment<wmma::matrix_a, M, N, K, wmma::precision::tf32, wmma::row_major> a[WARP_COL_TILES];
                wmma::fragment<wmma::matrix_b, M, N, K, wmma::precision::tf32, wmma::col_major> b[WARP_ROW_TILES];

#pragma unroll
                for (int i = 0; i < WARP_COL_TILES; i++) {
                    size_t shmem_idx_a = (warpId / BLOCK_ROW_WARPS) * M * BLOCK_ROW_WARPS + (i * M);
                    const float *tile_ptr = &shmem[shmem_idx_a][k_step * K];

                    wmma::load_matrix_sync(a[i], tile_ptr, K * CHUNK_K + SKEW_FLOAT);

#pragma unroll
                    for (int t = 0; t < a[i].num_elements; t++) {
                        a[i].x[t] = wmma::__float_to_tf32(a[i].x[t]);
                    }
#pragma unroll
                    for (int j = 0; j < WARP_ROW_TILES; j++) {
                        if (i == 0) {
                            // Load the B matrix fragment once, because it is going to be reused
                            // against the other A matrix fragments.
                            size_t shmem_idx_b = shmem_idx_b_off + (WARP_ROW_TILES * N) * (warpId%2) + (j * N);
                            const float *tile_ptr = &shmem[shmem_idx_b][k_step * K];

                            wmma::load_matrix_sync(b[j], tile_ptr, K * CHUNK_K + SKEW_FLOAT);
#pragma unroll
                            for (int t = 0; t < b[j].num_elements; t++) {
                                b[j].x[t] =  wmma::__float_to_tf32(b[j].x[t]);
                            }
                        }

                        wmma::mma_sync(c[i][j], a[i], b[j], c[i][j]);
                    }
                }
            }
            pipe.consumer_release();
            __syncthreads();
        }

        // Store the D fragments to shared memory.
#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
            for (int j = 0; j < WARP_ROW_TILES; j++) {
                float *tile_ptr = shmem_warp_tile_ptr + i * SHMEM_STRIDE * N + j * N;

                wmma::store_matrix_sync(tile_ptr, c[i][j], SHMEM_STRIDE, C_LAYOUT);
            }
        }

        __syncthreads();

        // Now that shared memory contains all the D tiles, stream them to global memory.
        float *dst_gmem_warp_stream_ptr = &D[gmem_idx];

#pragma unroll
        for (int i = 0; i < N; i++) {
            *((float4*)(dst_gmem_warp_stream_ptr + GLOBAL_MEM_STRIDE * i) + laneId) =
                *((float4*)(shmem_warp_stream_ptr + SHMEM_STRIDE * i) + laneId);
        }

        __syncthreads();
    }
#endif
}

void computeMM(const float *A, const float *B, float *C, int m, int k, int n)
{
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid((m+threads.x-1)/threads.x, (n+threads.y-1)/threads.y); 
    const int SHMEM_SZ = MAX(sizeof(float) * (BLOCK_COL_TILES * M) * (CHUNK_K * K + SKEW_FLOAT) * 2,
                       M * (BLOCK_ROW_WARPS * WARP_ROW_TILES) * N * (BLOCK_COL_WARPS * WARP_COL_TILES) * sizeof(float));
    gpuErrchk( hipFuncSetAttribute(reinterpret_cast<const void*>(compute_tf32gemm_async_copy), hipFuncAttributeMaxDynamicSharedMemorySize, SHMEM_SZ) );
    compute_tf32gemm_async_copy<<<deviceProp.multiProcessorCount*2, THREADS_PER_BLOCK, SHMEM_SZ>>>(A, B, C);
    //MatrixMulCUDA<<<grid, threads>>>(C, A, B, k, n);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    return;
}

